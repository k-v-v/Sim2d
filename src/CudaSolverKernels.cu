#include "hip/hip_runtime.h"
/*
 * CudaSolverKernels.cu
 *
 *  Created on: 10 Feb 2018
 *      Author: konstantin
 */
#include "CudaSolverKernels.h"
#include "Util.h"
#include "Constants.h"
//Compute F component. X and Y and block local

#define BLOCK_SIZE 16


//Device Constants
__constant__ float w;
__constant__ float reynold;
__constant__ float deltaTime;
__constant__ float cellSize;
__constant__ float gravityX;
__constant__ float gravityY;

/*
 * Optimisation notes:
 * In computeF, computeG if we unwrap uGrid(x, y) + deltaTime*((1.0f/reynold)*(duuxx+duuyy) - duux - duvy + gravityX);
 * and calculate each element and add it. It is possible to reduce the number of registers in use.
 * Don't know if this matters.
 * */
class Grid{
public:
	__device__ Grid(float * ptr, int size){
		ptr_ = ptr;
		size_ = size;
	}
	__device__ inline float& operator()(int x, int y){
		return ptr_[y * size_ + x];
	}
	__device__ inline const float& operator()(int x, int y)const{
		return ptr_[y * size_ + x];
	}
private:
	int size_;
	float * ptr_;
};
/**
	Copies a grid from global to local memory
	Warning: Should only be called on a thread which corresponds to a poin inside the grid
    @param global Pointer to global memory
    @param local Pointer to local memory block
    @param globalSize True Size of the edge the global grid
*/
//Note gridSize + 2 = globalSize
__device__ inline float fsquare(float val){
	return val*val;
}
__device__ inline float computeF(int x, int y, const Grid& uGrid, const Grid& vGrid){
	float duux = (1.0f/cellSize)*
			(
					fsquare(((uGrid(x, y) + uGrid(x+1, y) )/ 2.0f))
					-
					fsquare(((uGrid(x-1, y) + uGrid(x, y)) / 2.0f))
			)
			+
			GAMMA*(1.0f/cellSize)*
			(
					fabsf((uGrid(x, y)+uGrid(x+1, y))/2.0f)*((uGrid(x, y)-uGrid(x+1,y))/2.0f)
					-
					fabsf((uGrid(x-1, y)+uGrid(x, y))/2.0f)*((uGrid(x -1, y)-uGrid(x,y))/2.0f));


	float duvy = (1.0f/cellSize)*
			(
					((vGrid(x, y)+vGrid(x+1, y))/2.0f)*((uGrid(x, y)+uGrid(x,y+1))/2.0f)
					-
					((vGrid(x, y -1)+vGrid(x+1, y-1))/2.0f)*((uGrid(x, y -1)+uGrid(x,y))/2.0f)
			)
			+
			GAMMA*(1.0f/cellSize)*
			(
					fabsf((vGrid(x, y)+vGrid(x+1, y))/2.0f)*((uGrid(x, y)-uGrid(x,y+1))/2.0f)
					-
					fabsf((vGrid(x, y-1)+vGrid(x+1, y-1))/2.0f)*((uGrid(x, y-1)-uGrid(x,y))/2.0f)
			);

	float duuxx = 	(
					uGrid(x+1, y) - 2*uGrid(x, y) + uGrid(x-1, y)
					)
					/ fsquare(cellSize);
	float duuyy =	(
					uGrid(x, y+1) - 2*uGrid(x, y) + uGrid(x, y-1)
					)
					/ fsquare(cellSize);

	float F = uGrid(x, y) + deltaTime*((1.0f/reynold)*(duuxx+duuyy) - duux - duvy + gravityX);
	return F;
}
__device__ inline float computeG(int x, int y, const Grid& uGrid, const Grid& vGrid ){
	float duvx= (1.0f/cellSize)*
			(
					((uGrid(x,y)+uGrid(x,y+1))/2.0f)*((vGrid(x,y)+vGrid(x+1,y))/2.0f)
					-
					((uGrid(x-1,y)+uGrid(x-1,y+1))/2.0f)*((vGrid(x-1,y )+vGrid(x,y))/2.0f)
			)
			+
			GAMMA*(1.0f/cellSize)*
			(
					fabsf((uGrid(x,y)+uGrid(x,y+1))/2.0f)*((vGrid(x,y)-vGrid(x+1,y))/2.0f)
					-
					fabsf((uGrid(x-1,y)+uGrid(x-1,y+1))/2.0f)*((vGrid(x-1,y)-vGrid(x,y))/2.0f)
			);

	float dvvy=(1.0f/cellSize)*
			(
					fsquare(((vGrid(x,y)+vGrid(x,y+1))/2.0f))
					-
					fsquare(((vGrid(x,y-1)+vGrid(x,y))/2.0f))
			)
			+
			GAMMA*(1.0f/cellSize)*
			(
					fabsf((vGrid(x,y)+vGrid(x,y+1))/2.0f)*((vGrid(x,y)-vGrid(x,y+1))/2.0f)
					-
					fabsf((vGrid(x,y-1)+vGrid(x,y))/2.0f)*((vGrid(x,y-1)-vGrid(x,y))/2.0f)
			);

	float dvvxx = 	(
					vGrid(x+1, y) - 2*vGrid(x, y) + vGrid(x-1, y)
					)
					/ fsquare(cellSize);
	float dvvyy = 	(
					vGrid(x, y+1) - 2*vGrid(x, y) + vGrid(x, y-1)
					)
					/ fsquare(cellSize);


	float G = vGrid(x, y) + deltaTime*((1.0f/reynold)*(dvvxx+dvvyy) - dvvy - duvx + gravityY);

	return G;
}
__device__ inline float computeRHS(int x, int y, const Grid &fGrid, const Grid &gGrid){
	float rhs = (
					(fGrid(x,y)-fGrid(x-1,y))
					/
					cellSize
					+
					(gGrid(x,y)-gGrid(x,y-1))
					/
					cellSize)
				/deltaTime;
	return rhs;
}
__device__ inline float computeU(int x, int y, const Grid &fGrid, const Grid &pGrid){
	float u = fGrid(x, y)
			-
			(deltaTime/cellSize)
				*
			(pGrid(x+1,y) - pGrid(x,y));

	return u;
}
__device__ inline float computeV(int x, int y, const Grid &gGrid, const Grid &pGrid){
	float v = gGrid(x, y)
			-
			(deltaTime/cellSize)
				*
			(pGrid(x,y+1) - pGrid(x,y));

	return v;
}
__device__ inline float relax(int x, int y, const Grid pGrid, const Grid &rhsGrid){
	float cellsq = cellSize*cellSize;
	return
					(1-w)*pGrid(x,y)
						+
					w * (cellsq/4.0f)*
					(
							(pGrid(x+1,y)+pGrid(x-1,y))/cellsq
							+
							(pGrid(x,y+1)+pGrid(x,y-1))/cellsq
							-
							pGrid(x,y)
					);
	/*
	float residual = (
							(pg(x+1,y) - pg(x,y))
							-
							(pg(x,y) - pg(x-1,y))
						)/(cellsq)
						+
						(	(pg(x,y+1) - pg(x,y))
							-
							(pg(x,y) - pg(x,y-1))
						)/(cellsq)
						-
						rh(x,y);
	*/
}
__device__ inline void load_local_grid(float * global, float * local, int globalEdgeSize){
	int gridX = blockIdx.x * blockDim.x + threadIdx.x;	//Coordinates inside the kernel grid
	int gridY = blockIdx.y * blockDim.y + threadIdx.y;

	//Copy Global memory to correspoding local memory inisde the work group
	//How local memory should look (assumuing blockDim.x == blockDim.y == 8
	//Local memory should be 10 x 10 grid

	local[(threadIdx.y +1) * (blockDim.x + 2) + threadIdx.x + 1] = global[(gridY + 1)* globalEdgeSize  + gridX + 1];
	//Local memory now
	// 0 0 0 0 0 0 0 0 0 0
	// 0 X X X X X X X X 0
	// 0 X X X X X X X X 0
	// 0 X X X X X X X X 0
	// 0 X X X X X X X X 0
	// 0 X X X X X X X X 0
	// 0 X X X X X X X X 0
	// 0 X X X X X X X X 0
	// 0 X X X X X X X X 0
	// 0 0 0 0 0 0 0 0 0 0
	//Now copy memory from boundary conditions

	//----COPY BOUNDARY CONDITIONS
	if(threadIdx.x == 0 ){
		local[(threadIdx.y +1) * (blockDim.x + 2) ] = global[(gridY + 1)* globalEdgeSize  + gridX];
	}
	//Local memory now
	// 0 0 0 0 0 0 0 0 0 0
	// x X X X X X X X X 0
	// x X X X X X X X X 0
	// x X X X X X X X X 0
	// x X X X X X X X X 0
	// x X X X X X X X X 0
	// x X X X X X X X X 0
	// x X X X X X X X X 0
	// x X X X X X X X X 0
	// 0 0 0 0 0 0 0 0 0 0
	if(threadIdx.x == blockDim.x - 1 ){
		local[(threadIdx.y +1) * (blockDim.x + 2) + threadIdx.x + 2] = global[(gridY + 1)* globalEdgeSize  + gridX + 2];
	}
	//Local memory now
	// 0 0 0 0 0 0 0 0 0 0
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// 0 0 0 0 0 0 0 0 0 0
	if(threadIdx.y == 0 ){
		local[ threadIdx.x + 1] = global[(gridY)* globalEdgeSize  + gridX + 1];
	}
	//Local memory now
	// 0 x x x x x x x x 0
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// 0 0 0 0 0 0 0 0 0 0
	if(threadIdx.y == blockDim.y -1 ){
		local[(threadIdx.y + 2) * (blockDim.x + 2) + threadIdx.x + 1] = global[(gridY + 2)* globalEdgeSize  + gridX + 1];
	}
	//Local memory now
	// 0 x x x x x x x x 0
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// 0 x x x x x x x x 0
	if(threadIdx.x == 1 and threadIdx.y == 1){
		local[0] = global[(gridY - 1)* globalEdgeSize  + gridX - 1];
	}
	//Local memory now
	// X x x x x x x x x 0
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// 0 x x x x x x x x 0

	if(threadIdx.x == blockDim.x-2 and threadIdx.y == blockDim.y-2){
		local[(blockDim.y + 2) * (blockDim.x + 2) -1] = global[(gridY + 2)* globalEdgeSize  + gridX + 1];
	}
	//Local memory now
	// X x x x x x x x x 0
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// 0 x x x x x x x x x

	if(threadIdx.x == 1 and threadIdx.y == blockDim.y-2){
		local[(blockDim.y + 1) * (blockDim.x + 2)] = global[(gridY + 1)* globalEdgeSize];
	}
	//Local memory now
	// X x x x x x x x x 0
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x x x x x x x x x X

	if(threadIdx.x == blockDim.x-2 and threadIdx.y == 1){
		local[blockDim.x + 1] = global[globalEdgeSize - 1];
	}
	//Local memory now
	// X x x x x x x x x x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// x X X X X X X X X x
	// 0 x x x x x x x x x
	__syncthreads();
}
__global__ void clear_staggered_grid_kernel(float * UGrid,
											float * VGrid,
											float * FGrid,
											float * GGrid,
											float * RHSGrid,
											float * PressureGrid,
											int edgeSize){

	int gridX = blockIdx.x * blockDim.x + threadIdx.x;
	int gridY = blockIdx.y * blockDim.y + threadIdx.y;
	int location = gridY * edgeSize + gridX;
	if(location >= edgeSize*edgeSize){
		return ;
	}
	UGrid[gridY * edgeSize + gridX] = 0.0f;
	VGrid[gridY * edgeSize + gridX] = 0.0f;
	FGrid[gridY * edgeSize + gridX] = 0.0f;
	GGrid[gridY * edgeSize + gridX] = 0.0f;
	RHSGrid[gridY * edgeSize + gridX] = 0.0f;
	PressureGrid[gridY * edgeSize + gridX] = 0.0f;


}
__global__ void null_boundary_kernel(float * grid, int globalEdgeSize){
	//Note: kernel is 1 dismentional
	int gridX = blockIdx.x * blockDim.x + threadIdx.x;
	int location;

	location = gridX;
	if(location < globalEdgeSize * globalEdgeSize)
		grid[location] = 0.0f; 							//North boundary

	location = globalEdgeSize * (globalEdgeSize - 1) + gridX;
	if(location < globalEdgeSize * globalEdgeSize)
		grid[location] = 0.0f;							//South  boundary

	location = globalEdgeSize * gridX;
	if(location < globalEdgeSize*globalEdgeSize)
		grid[location] = 0.0f; 							//West  boundary

	location = globalEdgeSize * (gridX+1) - 1;
	if(location < globalEdgeSize * globalEdgeSize)
		grid[location] = 0.0f;							//East boundary
}
/* Grid size is the size of the non edge-grid (aka 2 smaller than gridSize)
 * We are executing for innerEdgeSize x innerEdgeSize grid
 * */
__global__ void computeFGGrid_kernel(float * uGridPtr, float * vGridPtr, float * fGridPtr, float * gGridPtr, int innerEdgeSize){
	//Shared mem should be size (blockDim.x + 2)^2
	extern __shared__ float shared_mem[];

	float * sharedUGrid = (float*) shared_mem;
	float * sharedVGrid = (float*) &sharedUGrid[(blockDim.x+2)*(blockDim.x+2)];

	int gridX = blockIdx.x * blockDim.x + threadIdx.x;	//Coordinates inside the kernel grid
	int gridY = blockIdx.y * blockDim.y + threadIdx.y;


	if(gridY > innerEdgeSize || gridX > innerEdgeSize){
		return;
	}

	load_local_grid(uGridPtr, sharedUGrid, innerEdgeSize + 2);
	load_local_grid(vGridPtr, sharedVGrid, innerEdgeSize + 2);

	Grid uGrid(sharedUGrid, blockDim.x + 2);
	Grid vGrid(sharedVGrid, blockDim.x + 2);

	int globalId = (gridY + 1)* (innerEdgeSize+2) + gridX + 1;

	//sharedUGrid and sharedVGrid should be (gridDim.x + 2)^2
	fGridPtr[globalId] = computeF( threadIdx.x + 1, threadIdx.y + 1, uGrid, vGrid);
	gGridPtr[globalId] = computeG( threadIdx.x + 1, threadIdx.y + 1, uGrid, vGrid);

}
//Copies the velocities from the edges of the ibber grid to the edges of the outer grid
__global__ void copyUVBoundaries_kernel(float *uGrid, float *vGrid, float * fGrid, float *gGrid, int globalEdgeSize){
	int gridX = blockIdx.x * blockDim.x + threadIdx.x;
	int location;

	//North boundary
	location = gridX;
	if(location < globalEdgeSize * globalEdgeSize){
		fGrid[location] = uGrid[location + globalEdgeSize];
		gGrid[location] = vGrid[location + globalEdgeSize];
	}
	//South  boundary
	location = globalEdgeSize * (globalEdgeSize - 1) + gridX;
	if(location < globalEdgeSize*globalEdgeSize){
		fGrid[location] = uGrid[location - globalEdgeSize];
		gGrid[location] = vGrid[location - globalEdgeSize];
	}
	//West  boundary
	location = globalEdgeSize * gridX;
	if(location < globalEdgeSize*globalEdgeSize){
		fGrid[location] = uGrid[location + 1];
		gGrid[location] = vGrid[location + 1];
	}
	//East boundary
	location = globalEdgeSize*(gridX+1) - 1;
	if(location < globalEdgeSize*globalEdgeSize){
		fGrid[location] = uGrid[location - 1];
		gGrid[location] = vGrid[location - 1];
	}
}
__global__ void copyPressureBoundaries(float *pGrid, int globalEdgeSize){
	int gridX = blockIdx.x * blockDim.x + threadIdx.x;
	int location;

	//North boundary
	location = gridX;
	if(location < globalEdgeSize * globalEdgeSize){
		pGrid[location] = pGrid[location + globalEdgeSize];
		pGrid[location] = pGrid[location + globalEdgeSize];
	}
	//South  boundary
	location = globalEdgeSize * (globalEdgeSize - 1) + gridX;
	if(location < globalEdgeSize*globalEdgeSize){
		pGrid[location] = pGrid[location - globalEdgeSize];
		pGrid[location] = pGrid[location - globalEdgeSize];
	}
	//West  boundary
	location = globalEdgeSize * gridX;
	if(location < globalEdgeSize*globalEdgeSize){
		pGrid[location] = pGrid[location + 1];
		pGrid[location] = pGrid[location + 1];
	}
	//East boundary
	location = globalEdgeSize*(gridX+1) - 1;
	if(location < globalEdgeSize*globalEdgeSize){
		pGrid[location] = pGrid[location - 1];
		pGrid[location] = pGrid[location - 1];
	}
}
__global__ void computeRHSGrid_kernel(float * fGridPtr, float * gGridPtr, float * rGridPtr, int innerEdgeSize){
	extern __shared__ float shared_mem[];

	float * sharedFGrid = (float*) shared_mem;
	float * sharedGGrid = (float*) &sharedFGrid[(blockDim.x+2)*(blockDim.x+2)];

	int gridX = blockIdx.x * blockDim.x + threadIdx.x;	//Coordinates inside the kernel grid
	int gridY = blockIdx.y * blockDim.y + threadIdx.y;


	if(gridY > innerEdgeSize || gridX > innerEdgeSize){
		return;
	}

	load_local_grid(fGridPtr, sharedFGrid, innerEdgeSize + 2);
	load_local_grid(gGridPtr, sharedGGrid, innerEdgeSize + 2);


	Grid fGrid(sharedFGrid, blockDim.x + 2);
	Grid gGrid(sharedGGrid, blockDim.x + 2);

	int globalId = (gridY + 1)* (innerEdgeSize+2) + (gridX + 1);

	//sharedUGrid and sharedVGrid should be (gridDim.x + 2)^2
	rGridPtr[globalId] = computeRHS(threadIdx.x + 1, threadIdx.y + 1, fGrid, gGrid);

}
__global__ void computeRedCells_kernel(float * pGridPtr, float * rhsGridPtr, int innerEdgeSize){
	extern __shared__ float pressure_cache[];			//For pressure grid only

	int gridX = blockIdx.x * blockDim.x + threadIdx.x;	//Coordinates inside the kernel grid
	int gridY = blockIdx.y * blockDim.y + threadIdx.y;
	if(gridX>=innerEdgeSize || gridY >= innerEdgeSize){
		return;
	}
	load_local_grid(pGridPtr, pressure_cache, innerEdgeSize + 2);
	//First copy all the black cells in a grid with 2x size of the block
	Grid pressureGrid(pressure_cache, blockDim.x + 2);
	Grid rhsGrid(rhsGridPtr, blockDim.x + 2);


	//If even

	if(!((gridY * innerEdgeSize + gridX) & 1)){
		int globalId = (gridY + 1)* (innerEdgeSize+2) + (gridX + 1);

		pGridPtr[globalId] = relax(threadIdx.x + 1, threadIdx.y + 1, pressureGrid, rhsGrid);
	}
	//Naive implementation
}
__global__ void computeBlackCells_kernel(float * pGridPtr, float * rhsGrdiPtr, int innerEdgeSize){
	extern __shared__ float pressure_cache[];			//For pressure grid only
	int gridX = blockIdx.x * blockDim.x + threadIdx.x;	//Coordinates inside the kernel grid
	int gridY = blockIdx.y * blockDim.y + threadIdx.y;

	if(gridX>=innerEdgeSize || gridY >= innerEdgeSize){
		return;
	}
	load_local_grid(pGridPtr, pressure_cache, innerEdgeSize + 2);
	//First copy all the black cells in a grid with 2x size of the block
	Grid pressureGrid(pressure_cache, blockDim.x + 2);
	Grid rhsGrid(rhsGrdiPtr, blockDim.x + 2);

	//If even

	if(((gridY * innerEdgeSize + gridX) & 1)){
		int globalId = (gridY + 1)* (innerEdgeSize+2) + (gridX + 1);
		pGridPtr[globalId] = relax(threadIdx.x + 1, threadIdx.y + 1, pressureGrid, rhsGrid);
	}
	//Naive implementation
}
__global__ void computeUV_kernel(float *pGridPtr, float* uGridPtr, float * vGridPtr, float * fGridPtr, float * gGridPtr  , int innerEdgeSize){
	extern __shared__ float pressure_cache[];			//For pressure grid only
	int gridX = blockIdx.x * blockDim.x + threadIdx.x;	//Coordinates inside the kernel grid
	int gridY = blockIdx.y * blockDim.y + threadIdx.y;

	if(gridX>=innerEdgeSize || gridY >= innerEdgeSize){
		return;
	}


	load_local_grid(pGridPtr, pressure_cache, innerEdgeSize + 2);
	//First copy all the black cells in a grid with 2x size of the block
	Grid pGrid(pressure_cache, blockDim.x + 2);
	Grid uGrid(uGridPtr, blockDim.x + 2);
	Grid vGrid(vGridPtr, blockDim.x + 2);

	Grid fGrid(fGridPtr, blockDim.x + 2);
	Grid gGrid(gGridPtr, blockDim.x + 2);

	//No point in making separete inline functions for simple computations so add +1 for easier indexing
	//Note: pGrid is in shared memory and all the other grids are global memory
	uGrid(gridX, gridY) = fGrid(gridX, gridY) -
					((deltaTime)/cellSize)*
					(
							pGrid(threadIdx.x+1,threadIdx.y)-pGrid(threadIdx.x,threadIdx.y)
					);

	vGrid(gridX, gridY) = gGrid(gridX,gridY) -
				((deltaTime)/cellSize)*
				(
						pGrid(threadIdx.x,threadIdx.y+1)-pGrid(threadIdx.x,threadIdx.y)
				);

}
__global__ void setGridPoint_kernel(float * grid, float value, int point){
	grid[point] = value;
}
//HOST FUNCTIONS

//Note this is the true edge size
void clear_staggered_grid_host( float * UGrid,
								float * VGrid,
								float * FGrid,
								float * GGrid,
								float * RHSGrid,
								float * PressureGrid,
								int globalEdgeSize){
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 numBlocks(globalEdgeSize / threadsPerBlock.x  + 1, globalEdgeSize / threadsPerBlock.y + 1, 1);
	clear_staggered_grid_kernel<<<numBlocks, threadsPerBlock>>>(UGrid, VGrid, FGrid, GGrid, RHSGrid, PressureGrid, globalEdgeSize);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
}
/*Sets the edges of the grid to null*/
void null_boundary_host(float * grid, int innerEdgeSize){
    dim3 threadsPerBlock(BLOCK_SIZE * BLOCK_SIZE, 1 , 1);
    dim3 numBlocks((innerEdgeSize+2) / threadsPerBlock.x  + 1, 1, 1);
    null_boundary_kernel<<<numBlocks, threadsPerBlock>>>(grid, (innerEdgeSize+2));

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
}
/*
 * Only runs on the cells not on the edges
 * */
void computeFGGrid_host(float * uGrid, float * vGrid, float * fGrid, float * gGrid, int innerEdgeSize){
	//Copy all the boundaries
    dim3 threadsPerBlock(BLOCK_SIZE * BLOCK_SIZE, 1 , 1);
    dim3 numBlocks((innerEdgeSize+2) / threadsPerBlock.x  + 1, 1, 1);
    copyUVBoundaries_kernel<<<numBlocks, threadsPerBlock>>>(uGrid, vGrid, fGrid, gGrid, innerEdgeSize + 2);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

    //Reuse threadsPerBlock and numBlocks
    threadsPerBlock.x = BLOCK_SIZE;
    threadsPerBlock.y = BLOCK_SIZE;
    numBlocks.x = innerEdgeSize / threadsPerBlock.x  + 1;
    numBlocks.y = innerEdgeSize / threadsPerBlock.y + 1;

    computeFGGrid_kernel
    <<<numBlocks, threadsPerBlock, 2*(BLOCK_SIZE+2)*(BLOCK_SIZE+2)*sizeof(float)>>>(uGrid, vGrid, fGrid, gGrid, innerEdgeSize);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
}
void computeRHSGrid_host(float *fGrid, float *gGrid, float * rGrid, int innerEdgeSize){
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 numBlocks(innerEdgeSize / threadsPerBlock.x  + 1, innerEdgeSize / threadsPerBlock.y + 1, 1);

	computeRHSGrid_kernel
	<<<numBlocks, threadsPerBlock, 2*(BLOCK_SIZE+2)*(BLOCK_SIZE+2)*sizeof(float)>>>(fGrid, gGrid, rGrid, innerEdgeSize);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
    //Compute RHS
}

void doSorStep_host(float * pGrid, float * rhsGrid, int innerEdgeSize){
	//Copy the edges of the inner grid to the edges of the outergrid
	//Aka set boundary conditions
    dim3 threadsPerBlock(BLOCK_SIZE * BLOCK_SIZE, 1 , 1);
    dim3 numBlocks((innerEdgeSize+2) / threadsPerBlock.x  + 1, 1, 1);


    //__global__ void copyPressureBoundaries(float *pGrid, int globalEdgeSize){

    copyPressureBoundaries<<<numBlocks, threadsPerBlock>>>(pGrid, innerEdgeSize + 2);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());



    //Reuse threadsPerBlock and numBlocks
    threadsPerBlock.x = BLOCK_SIZE;
    threadsPerBlock.y = BLOCK_SIZE;
    numBlocks.x = innerEdgeSize / threadsPerBlock.x  + 1;
    numBlocks.y = innerEdgeSize / threadsPerBlock.y + 1;

    //Do the red checkerboard
    computeRedCells_kernel
    <<<numBlocks, threadsPerBlock, (BLOCK_SIZE+2)*(BLOCK_SIZE+2)*sizeof(float)>>>(pGrid, rhsGrid, innerEdgeSize);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());


    //Do the red checkerboard
    computeBlackCells_kernel
    <<<numBlocks, threadsPerBlock, (BLOCK_SIZE+2)*(BLOCK_SIZE+2)*sizeof(float)>>>(pGrid, rhsGrid, innerEdgeSize);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
}
void computeUV_host(float *pGrid, float *uGrid, float * vGrid, float *fGrid, float *gGrid, int innerEdgeSize){
	//__global__ void computeUV_kernel(float *pGridPtr, float* uGridPtr, float * vGridPtr, float * fGridPtr, float * gGridPtr  , int innerEdgeSize){
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 numBlocks(innerEdgeSize / threadsPerBlock.x  + 1, innerEdgeSize / threadsPerBlock.y + 1, 1);

	computeUV_kernel
	<<<numBlocks, threadsPerBlock, (BLOCK_SIZE+2)*(BLOCK_SIZE+2)*sizeof(float)>>>(pGrid, uGrid, vGrid, fGrid, gGrid, innerEdgeSize);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());

}

void setGridPoint_host(float * grid, float value, int point){
	//Most inefficient way to set a single value(sadly there isn't any other way)
	//must launch a single thread kernel
	setGridPoint_kernel<<<1, 1>>>(grid, value, point);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
}
void devSetOmega(float omega_){
	hipMemcpyToSymbol(HIP_SYMBOL(w), &omega_, sizeof(float));
}
void devSetReynold(float reynold_){
	hipMemcpyToSymbol(HIP_SYMBOL(reynold), &reynold_, sizeof(float));
}
void devSetDeltaTime(float deltaTime_){
	hipMemcpyToSymbol(HIP_SYMBOL(deltaTime), &deltaTime_, sizeof(float));
}
void devSetCellSize(float cellSize_){
	hipMemcpyToSymbol(HIP_SYMBOL(cellSize), &cellSize_, sizeof(float));
}
void devSetGravity(float gravityX_, float gravityY_){
	hipMemcpyToSymbol(HIP_SYMBOL(gravityX), &gravityX_, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(gravityY), &gravityY_, sizeof(float));
}
